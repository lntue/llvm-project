
#include <hip/hip_runtime.h>
// Checks that cuda compilation does the right thing when passed -march.
// (Specifically, we want to pass it to host compilation, but not to device
// compilation or ptxas!)

// RUN: %clang -### --target=x86_64-linux-gnu -c \
// RUN: -nogpulib -nogpuinc -march=haswell %s 2>&1 | FileCheck %s
// RUN: %clang -### --target=x86_64-linux-gnu -c \
// RUN: -nogpulib -nogpuinc -march=haswell --cuda-gpu-arch=sm_52 %s 2>&1 | FileCheck %s

// CHECK: "-cc1"{{.*}} "-triple" "nvptx
// CHECK-SAME: "-target-cpu" "sm_52"

// CHECK: ptxas
// CHECK-SAME: "--gpu-name" "sm_52"

// CHECK: "-cc1"{{.*}} "-target-cpu" "haswell"
